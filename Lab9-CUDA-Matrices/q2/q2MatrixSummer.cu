/* 
    @Author:    3sne ( Mukur Panchani )
    @FileName:  q2MatrixSummer.cu
    @Task:      CUDA program compute sums of two matrices, using different parallelism techniques.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void addMatRowThreads(int *a, int *b, int *c, int m, int n) {
    
    int id = threadIdx.x;
    for ( int i = 0; i < n; i++ ) {
        int ind = id * n + i;
        c[ind] = a[ind] + b[ind];
    }

}

__global__ void addMatColThreads(int *a, int *b, int *c, int m, int n) {
    
    int id = threadIdx.x;
    for ( int i = 0; i < m; i++ ) {
        int ind = i * n + id;
        c[ind] = a[ind] + b[ind];
    }

}

__global__ void addMatElementThread(int *a, int *b, int *c, int m, int n) {

    int ci = threadIdx.x;
    int ri = threadIdx.y;
    int id = ri * m + ci;
    c[id] = a[id] + b[id]; 

}

int main() {
    int *matA, *matB, *matC;
    int *da, *db, *dc;
    int m, n;
    printf("== Enter Dimension of Matrix A and B (m x n) ==\n");
    printf("m >> "); scanf("%d", &m);
    printf("n >> "); scanf("%d", &n);

    matA = (int*)malloc(sizeof(int) * m * n);
    matB = (int*)malloc(sizeof(int) * m * n);
    matC = (int*)malloc(sizeof(int) * m * n);

    printf("== Matrix A Elements ==\n");
    for(int i = 0; i < m * n; i++) {
        scanf("%d", &matA[i]);
    }
    printf("== Matrix B Elements ==\n");
    for(int i = 0; i < m * n; i++) {
        scanf("%d", &matB[i]);
    }

    hipMalloc((void **) &da, sizeof(int) * m * n);
    hipMalloc((void **) &db, sizeof(int) * m * n);
    hipMalloc((void **) &dc, sizeof(int) * m * n);

    hipMemcpy(da, matA, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(db, matB, sizeof(int) * m * n, hipMemcpyHostToDevice);

    printf("\nChoose a degree of parallelism >> \n");
    printf("1. Thread handles row\n");
    printf("2. Thread handles column\n");
    printf("3. Thread handles element\nChoice >> ");
    int choice = 0;
    scanf("%d", &choice);
    dim3 block_conf (n, m);
    switch(choice) {
        case 1://Part A: 1 Thread handles 1 row >>
            printf("Chose: Thread handles row\n");
            addMatRowThreads<<<1,m>>>(da, db, dc, m, n);
            break;

        case 2://Part B: 1 Thread handles 1 column >>
            printf("Chose: Thread handles column\n");
            addMatColThreads<<<1,n>>>(da, db, dc, m, n);
            break;

        case 3://Part C: 1 Thread handles 1 element >>
            printf("Chose: Thread handles element\n");
            addMatElementThread<<<1, block_conf>>>(da, db, dc, m, n);
            break;

        default: 
            printf("Bad Option, exiting ...\n");
            exit(EXIT_FAILURE);
            break;
    }

    hipMemcpy(matC, dc, sizeof(int) * m * n, hipMemcpyDeviceToHost);
    printf("== Matrix C Elements (computed by choice %d)==\n", choice);
    for ( int i = 0; i < m; i++ ) {
        for ( int j = 0; j < n; j++ ) {
            printf("%d ", matC[i * n + j]);
        }
        printf("\n");
    }

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(matA);
    free(matB);
    free(matC);

    return 0;

}
/* 
    @Author:    3sne ( Mukur Panchani )
    @FileName:  q1BlockSizeN.cu
    @Task:      CUDA program to add two vectors parallely with N blocks but only 1 thread per block.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 
__global__ void add (int *a, int *b, int *c) {
    /*
        Adds Vectors a and b, puts result in c. 
    */
    int id = blockIdx.x;
    c[id] = a[id] + b[id];
}
 
int main() {
    int LS = 10240;
    int *a = (int*)malloc(sizeof(int) * LS);
    int *b = (int*)malloc(sizeof(int) * LS);
    int *c = (int*)malloc(sizeof(int) * LS);
    for (int i = 0; i < LS; i++ ) {
        a[i] = rand() % 99 + 1;
        b[i] = rand() % 99 + 1;
        c[i] = 0;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, sizeof(int) * LS);
    hipMalloc((void **)&d_b, sizeof(int) * LS);
    hipMalloc((void **)&d_c, sizeof(int) * LS);

    hipMemcpy(d_a, a, sizeof(int) * LS, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int) * LS, hipMemcpyHostToDevice);

    add<<<LS,1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, sizeof(int) * LS, hipMemcpyDeviceToHost);

    for (int i = 0; i < LS; i++ ) {
        printf("%4d  + %4d == %4d\n", a[i], b[i], c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
/* 
    @Author:    3sne ( Mukur Panchani )
    @FileName:  q3MatrixMul.cu
    @Task:      CUDA program computes product of two matrices, using different parallelism techniques.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void MatMulRowThreads(int *a, int *b, int *c, int m, int n, int q) {
    
    int id = threadIdx.x; //row id
    for ( int i = 0; i < q; i++ ) {
        c[id * q + i] = 0;
        for ( int k = 0; k < n; k++ ) {
            c[id * q + i] += a[id * n + k] * b[k * q + i];
        }
    }

}

__global__ void MatMulColThreads(int *a, int *b, int *c, int m, int n, int q) {
    
    int id = threadIdx.x; //column id
    for ( int i = 0; i < m; i++ ) {
        c[i * q + id] = 0;
        for ( int k = 0; k < n; k++ ) {
            c[i * q + id] += a[i * m + k] * b[k * q + id];
        }
    }

}
 
__global__ void MatMulElementThread(int *a, int *b, int *c, int m, int n, int q) {

    int ci = threadIdx.x;
    int ri = threadIdx.y;
    int id = ri * m + ci;
    c[id] = 0;
    for ( int k = 0; k < n; k++ ) {
        c[id] += a[ri * n + k] * b[k * q + ci];
    }
}
 
int main() {
    int *matA, *matB, *matC;
    int *da, *db, *dc;
    int m, n, p, q;
    printf("== Enter Dimension of Matrix A (m x n) ==\n");
    printf("m >> "); scanf("%d", &m);
    printf("n >> "); scanf("%d", &n);
    matA = (int*)malloc(sizeof(int) * m * n);
    printf("== Matrix A Elements ==\n");
    for(int i = 0; i < m * n; i++) {
        scanf("%d", &matA[i]);
    }
    
    printf("== Enter Dimension of Matrix B (p x q) ==\n");
    printf("p >> "); scanf("%d", &p);
    printf("q >> "); scanf("%d", &q);
    matB = (int*)malloc(sizeof(int) * p * q);
    if ( n != p ) {
        printf("[ERROR] n & p must be equal, Exiting ...\n");
        exit(EXIT_FAILURE);
    }
    printf("== Matrix B Elements ==\n");
    for(int i = 0; i < p * q; i++) {
        scanf("%d", &matB[i]);
    }

    matC = (int*)malloc(sizeof(int) * m * q);

    hipMalloc((void **) &da, sizeof(int) * m * n);
    hipMalloc((void **) &db, sizeof(int) * p * q);
    hipMalloc((void **) &dc, sizeof(int) * m * q);

    hipMemcpy(da, matA, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(db, matB, sizeof(int) * p * q, hipMemcpyHostToDevice);

    printf("\nChoose a degree of parallelism >> \n");
    printf("1. Thread handles row\n");
    printf("2. Thread handles column\n");
    printf("3. Thread handles element\nChoice >> ");
    int choice = 0;
    scanf("%d", &choice);
    dim3 block_conf (q, m);
    switch(choice) {
        case 1://Part A: 1 Thread handles 1 row >>
            printf("Chose: Thread handles row\n");
            MatMulRowThreads<<<1,m>>>(da, db, dc, m, n, q);
            break;

        case 2://Part B: 1 Thread handles 1 column >>
            printf("Chose: Thread handles column\n");
            MatMulColThreads<<<1,q>>>(da, db, dc, m, n, q);
            break;

        case 3://Part C: 1 Thread handles 1 element >>
            printf("Chose: Thread handles element\n");
            MatMulElementThread<<<1, block_conf>>>(da, db, dc, m, n, q);
            break;

        default: 
            printf("Bad Option, exiting ...\n");
            exit(EXIT_FAILURE);
            break;
    }

    hipMemcpy(matC, dc, sizeof(int) * m * n, hipMemcpyDeviceToHost);
    printf("== Matrix C Elements (computed by choice %d)==\n", choice);
    for ( int i = 0; i < m; i++ ) {
        for ( int j = 0; j < q; j++ ) {
            printf("%d ", matC[i * q + j]);
        }
        printf("\n");
    }

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(matA);
    free(matB);
    free(matC);

    return 0;
}
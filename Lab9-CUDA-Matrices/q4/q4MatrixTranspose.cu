/* 
    @Author:    3sne ( Mukur Panchani )
    @FileName:  q4MatrixTranspose.cu
    @Task:      CUDA program compute transpose of a matrix parallely.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void transpose(int *a, int *b, int m, int n) {
    /*
        Generates b: transpose of a
    */
    int ci = threadIdx.x;
    int ri = threadIdx.y;
    b[ci * m + ri] = a[ri * n + ci];
}

int main() {
    
    int *matA, *matB;
    int *da, *db;
    int m, n;
    printf("== Enter Dimension of Matrix A (m x n) ==\n");
    printf("m >> "); scanf("%d", &m);
    printf("n >> "); scanf("%d", &n);
    matA = (int*)malloc(sizeof(int) * m * n);
    matB = (int*)malloc(sizeof(int) * m * n);
    printf("== Matrix A Elements ==\n");
    for(int i = 0; i < m * n; i++) {
        scanf("%d", &matA[i]);
    }

    hipMalloc((void **)&da, sizeof(int) * m * n);
    hipMalloc((void **)&db, sizeof(int) * m * n);

    hipMemcpy(da, matA, sizeof(int) * m * n, hipMemcpyHostToDevice);
    dim3 block_conf (n, m);
    transpose<<<1, block_conf>>>(da, db, m, n);
    hipMemcpy(matB, db, sizeof(int) * m * n, hipMemcpyDeviceToHost);

    printf("== Matrix B Elements ==\n");
    for ( int i = 0; i < n; i++ ) {
        for ( int j = 0; j < m; j++ ) {
            printf("%d ", matB[i * m + j]);
        }
        printf("\n");
    }

    hipFree(da);
    hipFree(db);
    free(matA);
    free(matB);
    return 0;
}
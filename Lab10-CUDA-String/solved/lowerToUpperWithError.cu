#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define n 1024


__global__ void sc(char *a, char c[n]) {
    int i = threadIdx.x;
    c[i] = (char)((int)a[i] - 32);
    // printf("%s\n", c[i]);
}

int main() {
    char a[n], c[n], *pa, *pc;
    for (int i = 0; i < n; i++) {
        a[i] = 'a';
    }
    printf("C == \n");
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    
    hipMalloc((void**)&pa, n * sizeof(char));
    hipMalloc((void**)&pc, n * sizeof(char));
    hipMemcpy(pa, a, n * sizeof(char), hipMemcpyHostToDevice);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Err1: %s\n", hipGetErrorString(err));
    }

    sc<<<1, n>>>(pa, pc);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Err2: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(c, pc, n * sizeof(char), hipMemcpyDeviceToHost);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float et;
    hipEventElapsedTime(&et, start, stop);

    for (int i = 0; i < n; i++) {
        printf("%c -> %c\n", a[i], c[i]);
    }
    printf("Kernel exec time on device : %f\n", et);
    hipFree(pa);
    hipFree(pc);

}
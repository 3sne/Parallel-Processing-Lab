/* 
    @Author:    3sne ( Mukur Panchani )
    @FileName:  q1FindSubstring.cu
    @Task:      CUDA program that finds a substring in a given string.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void resetBuf(char* b, int blen) {
    for ( int i = 0; i < blen; i++ )
        b[i] = '\0';
}

__global__ void findSs(char *text, char *ss, int *retArr, int ssLen) {
    int tid = threadIdx.x;
    int w = 0;
    retArr[tid] = 1; 
    for (int i = tid; i < (tid + ssLen); i++ ) {
        if (text[i] != ss[w]) {
            retArr[tid] = 0;
            break;
        }
        w += 1;
    }
}

int main() {
    char *buf = (char*)malloc(sizeof(char) * 10240);
    char *text, *subStr, *dtext, *dsubStr;
    int  *retArr, *dretarr;

    printf("[IN] Enter Text >> ");
    scanf("%[^\n]s", buf);
    int tLen = strlen(buf);
    text = (char*)malloc(sizeof(char) * tLen);
    strcpy(text, buf);

    resetBuf(buf, 10240);

    printf("[IN] Enter Sub-String >> ");
    scanf("%s", buf);
    int ssLen = strlen(buf);
    subStr = (char*)malloc(sizeof(char) * ssLen);
    strcpy(subStr, buf);
    free(buf);

    retArr = (int*)malloc(sizeof(int) * (tLen - ssLen + 1));
    for (int i = 0; i < (tLen - ssLen + 1); i++) {
        retArr[i] = 0;
    }

    hipMalloc((void **)&dtext, sizeof(char) * tLen);
    hipMalloc((void **)&dsubStr, sizeof(char) * ssLen);
    hipMalloc((void **)&dretarr, sizeof(int) * (tLen - ssLen + 1));

    hipMemcpy(dtext, text, sizeof(char) * tLen, hipMemcpyHostToDevice);
    hipMemcpy(dsubStr, subStr, sizeof(char) * ssLen, hipMemcpyHostToDevice);
    hipMemcpy(dretarr, retArr, sizeof(int) * (tLen - ssLen + 1), hipMemcpyHostToDevice);
    dim3 block_conf (tLen - ssLen + 1, 1);
    findSs<<<1, block_conf>>>(dtext, dsubStr, dretarr, ssLen);
    hipMemcpy(retArr, dretarr, sizeof(int) * (tLen - ssLen + 1), hipMemcpyDeviceToHost);
    
    int yay = 0;
    for (int i = 0; i < tLen - ssLen + 1; i++) {
        if (retArr[i] == 1) {
            yay = 1;
            break;
        }
    }

    if (yay) {
        printf("Substring found in Text @ index(es) ");
        for (int i = 0; i < tLen - ssLen + 1; i++) {
            if (retArr[i])
                printf("%d ", i);
        }
    } else {
        printf("Substring un-found in Text :(");
    }
    printf("\n");

    hipFree(dtext);
    hipFree(dsubStr);
    hipFree(dretarr);
    free(text);
    free(subStr);
    return 0;
}
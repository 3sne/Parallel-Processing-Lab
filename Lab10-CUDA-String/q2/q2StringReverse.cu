/* 
    @Author:    3sne ( Mukur Panchani )
    @FileName:  q2StringReverse.cu
    @Task:      CUDA program that the reverses given string.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void reverse(char *str, char *rev, int len) {
	int tid = threadIdx.x;
	rev[len - tid - 1] = str[tid];
}

int main() {
	char *dstr, *drev;
	char str[256], rev[256];
	printf("Enter the string >> ");
	scanf("%s", str);
	int len = strlen(str);
	
	hipMalloc((void **)&dstr, len * sizeof(char));
	hipMalloc((void **)&drev, len * sizeof(char));
	
	hipMemcpy(dstr, str, len * sizeof(char), hipMemcpyHostToDevice);
	reverse<<<1, len>>>(dstr, drev, len);
	hipMemcpy(rev, drev, len * sizeof(char), hipMemcpyDeviceToHost);

	printf("Reverse: %s\n", rev);
}

/* 
    @Author:    3sne ( Mukur Panchani )
    @FileName:  q4MatAddAndMul2D.cu
    @Task:      CUDA program that calculates multiplication and addition of two matrices using 2D Grid & 2D Block.
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

__device__ int getTid() {
    int blockSkip = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y); 
    int rowSkip = (threadIdx.y * gridDim.x * blockDim.x);
    int rowDisplacement = (blockIdx.x * blockDim.x) + threadIdx.x;
    int tid = blockSkip + rowSkip + rowDisplacement;
    return tid;
}

__global__ void MatAddElementThread(int *a, int *b, int *d) {
    int tid = getTid();
    d[tid] = a[tid] + b[tid];
}

__global__ void MatMulElementThread(int *a, int *b, int *c, int n, int q) {
    int tid = getTid();
    int initDisp = tid % q;
    c[tid] = 0;
    for (int k = 0; k < n; k++) {
        c[tid] += a[tid - initDisp + k] * b[k * q + initDisp];
    }
}

int main() {
    system("clear");

    int *matA, *matB, *matProd, *matSum;
    int *da, *db, *dc, *dd;
    int m, n, p, q;
    int willMul = 1, willAdd = 1;
    
    printf("[NOTE] Both Matrices should have SAME and EVEN dimensions to successfully compute both the sum and the product (i.e, EVEN SQUARE MATRICES)\n");
    printf("\n== Enter Dimension of Matrix A (m x n) ==\n");
    printf("m >> "); scanf("%d", &m);
    printf("n >> "); scanf("%d", &n);
    matA = (int*)malloc(sizeof(int) * m * n);
    printf("== Matrix A Elements ==\n");
    for(int i = 0; i < m * n; i++) {
        scanf("%d", &matA[i]);
    }

    printf("\n== Enter Dimension of Matrix B (p x q) ==\n");
    printf("p >> "); scanf("%d", &p);
    printf("q >> "); scanf("%d", &q);
    if ( m % 2 || n % 2 || p % 2 || q % 2) {
        free(matA);
        printf("[PROGRAM] You don't listen to me, do you? I told you O N L Y  E V E N dimensions.....\n"); sleep(2);
        printf("[PROGRAM] Why this restriction? Because I use a fixed block size of (2, 2). For odd dimensions, I don't like keeping some threads lonely. Thread lives matter. /\n"); sleep(2);
        printf("[PROGRAM] But you clearly don't care, so don't I. Byeee Noob \\(^.^)\n"); sleep(2);
        exit(EXIT_FAILURE);
    }
    if (n != p) {
        willMul = 0;
        printf("[MUL ERROR] n & p must be equal, Skipping Matrix Multiplication...\n"); sleep(1);
    }
    if (m != p || n != q) {
        willAdd = 0;
        printf("[ADD ERROR] Dimensions of matA and matB are unequal, skipping Matrix Addition...\n"); sleep(1);
    }
    matB = (int*)malloc(sizeof(int) * p * q);
    printf("== Matrix B Elements ==\n");
    for(int i = 0; i < p * q; i++) {
        scanf("%d", &matB[i]);
    }

    matProd = (int*)malloc(sizeof(int) * m * q);
    matSum  = (int*)malloc(sizeof(int) * m * n);

    hipMalloc((void **) &da, sizeof(int) * m * n);
    hipMalloc((void **) &db, sizeof(int) * p * q);
    hipMalloc((void **) &dc, sizeof(int) * m * q);
    hipMalloc((void **) &dd, sizeof(int) * m * n);

    hipMemcpy(da, matA, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(db, matB, sizeof(int) * p * q, hipMemcpyHostToDevice);

    dim3 grid_conf  (q / 2, m / 2);
    dim3 block_conf (2, 2);
    
    if (willMul) {   
        MatMulElementThread<<<grid_conf, block_conf>>>(da, db, dc, n, q);
        hipMemcpy(matProd, dc, sizeof(int) * m * q, hipMemcpyDeviceToHost);
        printf("\n-=Result of Multiplication=-\n");
        printf("----------------------------\n");
        for (int i = 0; i < m; i++ ) {
            for (int j = 0; j < q; j++) {
                printf("%6d ", matProd[i * q + j]);
            }
            printf("\n");
        }
    }

    if (willAdd) {
        MatAddElementThread<<<grid_conf, block_conf>>>(da, db, dd);
        hipMemcpy(matSum, dd, sizeof(int) * m * n, hipMemcpyDeviceToHost);
        printf("\n-=Result of Addition=-\n");
        printf("----------------------\n");
        for (int i = 0; i < m; i++ ) {
            for (int j = 0; j < n; j++) {
                printf("%6d ", matSum[i * n + j]);
            }
            printf("\n");
        }
    }
    
    if (!willAdd && !willMul) {
        printf("Bad Matrix dimensions, exiting...\n");
    }

    printf("\n");
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
    free(matA);
    free(matB);
    free(matProd);
    free(matSum);
    return 0;
}
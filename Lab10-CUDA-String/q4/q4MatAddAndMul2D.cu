#include "hip/hip_runtime.h"
/* 
    @Author:    3sne ( Mukur Panchani )
    @FileName:  q4MatAddAndMul2D.cu
    @Task:      CUDA program that calculates multiplication and addition of two matrices using 2D Grid & 2D Block.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ int getTid() {
	int blockSkip = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y); 
	int rowSkip = (threadIdx.y * gridDim.x * blockDim.x);
	int rowDisplacement = (blockIdx.x * blockDim.x) + threadIdx.x;
	int tid = blockSkip + rowSkip + rowDisplacement;
	return tid;
}

__global__ void MatMulElementThread(int *a, int *b, int *c, int q) {
	int tid = getTid();
	int initDisp = tid % q;
	c[tid] = 0;
	for (int k = 0; k < n; k++) {
		c[tid] += a[tid - initDisp + k] * b[k * q + initDisp];
	}
}

int main() {
	int *matA, *matB, *matC;
	int *da, *db, *dc;
	int m, n, p, q;
	printf("ALL DIMENSIONS MUST BE EVEN\n");
	printf("== Enter Dimension of Matrix A (m x n) ==\n");
	printf("m >> "); scanf("%d", &m);
	printf("n >> "); scanf("%d", &n);
	matA = (int*)malloc(sizeof(int) * m * n);
	printf("== Matrix A Elements ==\n");
	for(int i = 0; i < m * n; i++) {
		scanf("%d", &matA[i]);
	}

	printf("== Enter Dimension of Matrix B (p x q) ==\n");
	printf("p >> "); scanf("%d", &p);
	printf("q >> "); scanf("%d", &q);
	matB = (int*)malloc(sizeof(int) * p * q);
	if ( n != p ) {
		printf("[ERROR] n & p must be equal, Exiting ...\n");
		exit(EXIT_FAILURE);
	}
	printf("== Matrix B Elements ==\n");
	for(int i = 0; i < p * q; i++) {
		scanf("%d", &matB[i]);
	}

	matC = (int*)malloc(sizeof(int) * m * q);

	hipMalloc((void **) &da, sizeof(int) * m * n);
	hipMalloc((void **) &db, sizeof(int) * p * q);
	hipMalloc((void **) &dc, sizeof(int) * m * q);

	hipMemcpy(da, matA, sizeof(int) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(db, matB, sizeof(int) * p * q, hipMemcpyHostToDevice);

	dim3 grid_conf  (q / 2, m / 2);
	dim3 block_conf (2, 2);
	MatMulElementThread<<<grid_conf, block_conf>>>(da, db, dc, q);
	hipMemcpy(matC, dc, sizeof(int) * m * q, hipMemcpyDeviceToHost);

	for (int i = 0; i < m; i++ ) {
		for (int j = 0; j < q; j++) {
			printf("%3d ", matC[i * q + j]);
		}
		printf("\n");
	}

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(matA);
	free(matB);
	free(matC);
	return 0;
}
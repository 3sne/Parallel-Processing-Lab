/* 
    @Author:    3sne ( Mukur Panchani )
    @FileName:  q3BinaryMaker.cu
    @Task:      CUDA program that converts chars of a string to binary.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void makeBinaries(char *str, int *binArr) {
	int tid = threadIdx.x;
	int n = (int)str[tid];
	int rem, i = 1, bin = 0;
    while (n > 0) {
		rem = n % 2;
		n = n / 2;
        bin = bin + rem * i;
        i = i * 10;
    }
    binArr[tid] = bin;
}	

int main() {
	char *dStr; int *dBinArr;
	char *str = (char*)malloc(sizeof(char) * 10240);
	printf("Enter the string >> "); scanf("%[^\n]s",  str);
	int len = strlen(str);
	int *binArr = (int*)malloc(sizeof(int) * len);
	
	hipMalloc((void **)&dStr , len*sizeof(char));
	hipMalloc((void **)&dBinArr, len*sizeof(int));
	
	hipMemcpy(dStr, str, len * sizeof(char), hipMemcpyHostToDevice);
	makeBinaries<<<1, len>>>( dStr,  dBinArr);
	hipMemcpy(binArr, dBinArr, len * sizeof(int), hipMemcpyDeviceToHost);

	printf("Output: \n");
	for(int i = 0 ; i < len; i++) {
		printf("'%c' -> %10d\n",  str[i] ,  binArr[i]);
	}
}
